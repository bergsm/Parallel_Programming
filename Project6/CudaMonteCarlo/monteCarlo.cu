#include "hip/hip_runtime.h"
// Array multiplication: C = A * B:

// System includes
#include <stdio.h>
#include <assert.h>
#include <malloc.h>
#include <math.h>
#include <stdlib.h>

// CUDA runtime
#include <hip/hip_runtime.h>

// Helper functions and utilities to work with CUDA
#include "helper_functions.h"
#include "hip/hip_runtime_api.h"

// for printing probability
#ifndef LAST
#define LAST        0
#endif

#ifndef BLOCKSIZE
#define BLOCKSIZE		32		// number of threads per block
#endif

#ifndef SIZE
#define SIZE			1*1024*1024	// array size
#endif

#ifndef NUMTRIES
#define NUMTRIES		100		// to make the timing more accurate
#endif

#ifndef TOLERANCE
#define TOLERANCE		0.00001f	// tolerance to relative error
#endif

// ranges for the random numbers:
const float XCMIN =  0.0;
const float XCMAX =  2.0;
const float YCMIN =  0.0;
const float YCMAX =  2.0;
const float RMIN  =  0.5;
const float RMAX  =  2.0;

// function prototypes:
float       Ranf( float, float );
int     Ranf( int, int );
void        TimeOfDaySeed( );


// array multiplication (CUDA Kernel) on the device: C = A * B

__global__  void MonteCarlo( float *xcs, float *ycs, float *rs , int *hits)
{
	//__shared__ float prods[BLOCKSIZE];

	//unsigned int numItems = blockDim.x;
	//unsigned int tnum = threadIdx.x;
	//unsigned int wgNum = blockIdx.x;
	unsigned int gid = blockIdx.x*blockDim.x + threadIdx.x;

    // grab location and radius of circle
    float xc = xcs[gid];
    float yc = ycs[gid];
    float r  = rs[gid];

    // solve for the intersection using the quadratic formula:
    float a = 2.;
    float b = -2.*( xc + yc );
    float c = xc*xc + yc*yc - r*r;
    float d = b*b - 4.*a*c;

    //If d is less than 0., then the circle completely missed. (Case A) Continue on to the next trail in the for-loop.
    if (d > 0)
    {
        // hits the circle:
        // get the first intersection:
        d = sqrt( d );
        float t1 = (-b + d ) / ( 2.*a );    // time to intersect the circle
        float t2 = (-b - d ) / ( 2.*a );    // time to intersect the circle
        float tmin = t1 < t2 ? t1 : t2;     // only care about the first intersection
        //If tmin is less than 0., then the circle completely engulfs the laser pointer. (Case B) Continue on to the next trial in the for-loop.
        if (tmin > 0)
        {

            // where does it intersect the circle?
            float xcir = tmin;
            float ycir = tmin;

            // get the unitized normal vector at the point of intersection:
            float nx = xcir - xc;
            float ny = ycir - yc;
            float n = sqrt( nx*nx + ny*ny );
            nx /= n;    // unit vector
            ny /= n;    // unit vector

            // get the unitized incoming vector:
            float inx = xcir - 0.;
            float iny = ycir - 0.;
            float in = sqrt( inx*inx + iny*iny );
            inx /= in;  // unit vector
            iny /= in;  // unit vector

            // get the outgoing (bounced) vector:
            float dot = inx*nx + iny*ny;
            float outx = inx - 2.*nx*dot;   // angle of reflection = angle of incidence`
            float outy = iny - 2.*ny*dot;   // angle of reflection = angle of incidence`

            // find out if it hits the infinite plate:
            float t = ( 0. - ycir ) / outy;
            //If t is less than 0., then the reflected beam went up instead of down. Continue on to the next trial in the for-loop.
            if (t > 0)
            {

        //Otherwise, this beam hit the infinite plate. (Case D) Increment the number of hits and continue on to the next trial in the for-loop.
                hits[gid] = 1;
            }
        }
    }



	//prods[tnum] = A[gid] * B[gid];

	//for (int offset = 1; offset < numItems; offset *= 2)
	//{
	//	int mask = 2 * offset - 1;
	//	__syncthreads();
	//	if ((tnum & mask) == 0)
	//	{
	//		prods[tnum] += prods[tnum + offset];
	//	}
	//}

	//__syncthreads();
	//if (tnum == 0)
	//	C[wgNum] = prods[0];
}


// main program:

int
main( int argc, char* argv[ ] )
{
    // print relevant information to screen
    fprintf(stderr, "Blocksize: %d\tArray size: %d\tTries: %d\n", BLOCKSIZE, SIZE, NUMTRIES);

    TimeOfDaySeed( );       // seed the random number generator

    // uncomment to print device information
	//int dev = findCudaDevice(argc, (const char **)argv);

	// allocate host memory:
    // better to define these here so that the rand() calls don't get into the thread timing:
    float *hxcs = new float [SIZE];
    float *hycs = new float [SIZE];
    float * hrs = new float [SIZE];
    int   *hits = new int [SIZE];

    // fill the random-value arrays:
    for( int n = 0; n < SIZE; n++ )
    {
            hxcs[n] = Ranf( XCMIN, XCMAX );
            hycs[n] = Ranf( YCMIN, YCMAX );
            hrs[n] = Ranf(  RMIN,  RMAX );
            hits[n] = 0;
    }

	//float * hA = new float [ SIZE ];
	//float * hB = new float [ SIZE ];
	//float * hC = new float [ SIZE/BLOCKSIZE ];

	//for( int i = 0; i < SIZE; i++ )
	//{
	//	hA[i] = hB[i] = (float) sqrt(  (float)(i+1)  );
	//}

	// allocate device memory:

	float *dA, *dB, *dC;
	int *dD;

	dim3 dimsA( SIZE, 1, 1 );
	dim3 dimsB( SIZE, 1, 1 );
	dim3 dimsC( SIZE, 1, 1 );
	dim3 dimsD( SIZE, 1, 1 );
//	dim3 dimsC( SIZE/BLOCKSIZE, 1, 1 );

	//__shared__ float prods[SIZE/BLOCKSIZE];


	hipError_t status;
	status = hipMalloc( reinterpret_cast<void **>(&dA), SIZE*sizeof(float) );
		checkCudaErrors( status );
	status = hipMalloc( reinterpret_cast<void **>(&dB), SIZE*sizeof(float) );
		checkCudaErrors( status );
	status = hipMalloc( reinterpret_cast<void **>(&dC), SIZE*sizeof(float) );
		checkCudaErrors( status );
	status = hipMalloc( reinterpret_cast<void **>(&dD), SIZE*sizeof(int) );
		checkCudaErrors( status );
	//status = hipMalloc( reinterpret_cast<void **>(&dC), (SIZE/BLOCKSIZE)*sizeof(float) );
		//checkCudaErrors( status );


	// copy host memory to the device:

	status = hipMemcpy( dA, hxcs, SIZE*sizeof(float), hipMemcpyHostToDevice );
		checkCudaErrors( status );
	status = hipMemcpy( dB, hycs, SIZE*sizeof(float), hipMemcpyHostToDevice );
		checkCudaErrors( status );
	status = hipMemcpy( dC, hrs, SIZE*sizeof(float), hipMemcpyHostToDevice );
		checkCudaErrors( status );
	status = hipMemcpy( dD, hits, SIZE*sizeof(int), hipMemcpyHostToDevice );
		checkCudaErrors( status );

	// setup the execution parameters:

	dim3 threads(BLOCKSIZE, 1, 1 );
	dim3 grid( SIZE / threads.x, 1, 1 );

	// Create and start timer

	hipDeviceSynchronize( );

	// allocate CUDA events that we'll use for timing:

	hipEvent_t start, stop;
	status = hipEventCreate( &start );
		checkCudaErrors( status );
	status = hipEventCreate( &stop );
		checkCudaErrors( status );

	// record the start event:

	status = hipEventRecord( start, NULL );
		checkCudaErrors( status );

	// execute the kernel:

	for( int t = 0; t < NUMTRIES; t++)
	{
	        MonteCarlo<<< grid, threads >>>( dA, dB, dC, dD );
	}

	// record the stop event:

	status = hipEventRecord( stop, NULL );
		checkCudaErrors( status );

	// wait for the stop event to complete:

	status = hipEventSynchronize( stop );
		checkCudaErrors( status );

	float msecTotal = 0.0f;
	status = hipEventElapsedTime( &msecTotal, start, stop );
		checkCudaErrors( status );

	// compute and print the performance

	double secondsTotal = 0.001 * (double)msecTotal;
	double multsPerSecond = (float)SIZE * (float)NUMTRIES / secondsTotal;
	double megaMultsPerSecond = multsPerSecond / 1000000.;
	fprintf( stderr, "Array Size = %10d, MegaTrials/Second = %10.2lf\n", SIZE, megaMultsPerSecond );
	printf("%10.2lf,", megaMultsPerSecond );

	// copy result from the device to the host:
	status = hipMemcpy( hits, dD, SIZE*sizeof(float), hipMemcpyDeviceToHost );
		checkCudaErrors( status );

    // deduce probability from hits array
    int totalHits = 0;
	float prob = 0.;
	for(int i = 0; i < SIZE; i++ )
	{
		totalHits += hits[i];
	}
    prob = (float)totalHits/SIZE;
	fprintf( stderr, "\nprobability = %4.4lf\n", prob );

    //print results
    if (LAST)
    {
        FILE* fp;
        fp = fopen("prob.txt", "w+");
        fprintf(fp, "%4.4lf", prob);
        fclose(fp);
    }

	// clean up memory:
	delete [ ] hxcs;
	delete [ ] hycs;
	delete [ ] hrs;
	delete [ ] hits;

	status = hipFree( dA );
		checkCudaErrors( status );
	status = hipFree( dB );
		checkCudaErrors( status );
	status = hipFree( dC );
		checkCudaErrors( status );
	status = hipFree( dD );
		checkCudaErrors( status );


	return 0;
}

float Ranf( float low, float high )
{
        float r = (float) rand();               // 0 - RAND_MAX
        float t = r  /  (float) RAND_MAX;       // 0. - 1.

        return   low  +  t * ( high - low );
}

int Ranf( int ilow, int ihigh )
{
        float low = (float)ilow;
        float high = ceil( (float)ihigh );

        return (int) Ranf(low,high);
}

void TimeOfDaySeed( )
{
    struct tm y2k = { 0 };
    y2k.tm_hour = 0;   y2k.tm_min = 0; y2k.tm_sec = 0;
    y2k.tm_year = 100; y2k.tm_mon = 0; y2k.tm_mday = 1;

    time_t  timer;
    time( &timer );
    double seconds = difftime( timer, mktime(&y2k) );
    unsigned int seed = (unsigned int)( 1000.*seconds );    // milliseconds
    srand( seed );
}


